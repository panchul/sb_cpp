#include "hip/hip_runtime.h"
// originated from NVidia's sample for vectorAdd

#include <stdio.h>

//#define MYTHREADS	1920
#define MYTHREADS	1024

#define MYELEMENTS	70000000

// Makefile passes it in via -DDO_CUDA
//#define DO_CUDA

//#ifdef DO_CUDA
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
//#include <hip/hip_runtime_api.h>
#endif

__device__
void myfunc1(float a, float b, float* c) {
        *(c) =
                (a + b)*(a + b)*(a+b)
                +(a + b*a + b)*(a+b)
                +(a * b)*(a + b*(a+b))
                +(a + b)*(a + b)*(a+b)
                +(a + b*(a + b)*a+b)
                +(a * b*(a - b))*(a+b)
                ;
}

#define myfunc(a, b, c) \
        * c = \
                (a + b)*(a + b)*(a+b) \
                +(a + b*a + b)*(a+b) \
                +(a * b)*(a + b*(a+b)) \
                +(a + b)*(a + b)*(a+b) \
                +(a + b*(a + b)*a+b) \
                +(a * b*(a - b))*(a+b) \

#ifdef DO_CUDA
/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void
vectorAdd(const float *A, const float *B, float *C, int numElements) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	//int j = 0;
	float *pC;
	if ( // j++ < MYTHREADS &&
		i < numElements) {
    		pC = &(C[i]);
        	myfunc1(A[i], B[i], pC);
	}
}
#else
void vectorAdd(const float *A, const float *B, float *C, int numElements) {
    int i = 0;
	float *pC;
    while( i < numElements) {
		pC = &(C[i]);
      	myfunc(A[i], B[i], pC);
		++i;
    }
}
#endif

/**
 * Host main routine
 */
int main(void) {

#ifdef DO_CUDA
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
#endif

    // Print the vector length to be used, and compute its size
    int numElements = MYELEMENTS;
    size_t size = numElements * sizeof(float);
    printf("[Vector addition of %d elements]\n", numElements);

    // Allocate the host input vector A
    float *h_A = (float *)malloc(size);

    // Allocate the host input vector B
    float *h_B = (float *)malloc(size);

    // Allocate the host output vector C
    float *h_C = (float *)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL) {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i) {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

#ifdef DO_CUDA
    // Allocate the device input vector A
    float *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector B
    float *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector C
    float *d_C = NULL;
    err = hipMalloc((void **)&d_C, size);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launch the Vector Add CUDA Kernel
    //int threadsPerBlock = 256;
    //int threadsPerBlock = 1;
    int threadsPerBlock = MYTHREADS;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    err = hipGetLastError();

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
#else
    vectorAdd(h_A, h_B, h_C, numElements);
#endif

    // Verify that the result vector is correct

	float myc;
	float* pmyc = &myc;
    for (int i = 0; i < numElements; ++i) {
	myfunc(h_A[i],h_B[i],pmyc);
        if (fabs(myc - h_C[i]) > 1e-5) {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    printf("Test PASSED\n");

#ifdef DO_CUDA
    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to free device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_B);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to free device vector B (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_C);

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to free device vector C (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
#endif

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    printf("Done\n");
    return 0;
}
