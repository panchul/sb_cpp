#include "hip/hip_runtime.h"
//
// nvcc list_threads.cu
// 
// basic into to cuda kernel
//

#include <cstdlib>
#include <iostream>

__global__
void saveTid(int *tids) {
    int tid = threadIdx.x;
    tids[tid] tid;
}

int main(int argc, char *argv[]) {

    int numThreads = atoi(argv[1]);
    int *dTids;
    
    hipMalloc(&tids, sizeof(int)*numThreads);
    saveTid<<< 1, numThreads >>>(dTids);
    
    int *hTids = new int[numThreads];
    
    hipMemcpy(hTids, dTids, sizeof(int) *numThreads, hipMemcpyDeviceToHost);

    for(int i = 0; i < numThreads; ++i) {
        std::cout << i << " - " << hTids[i] << "\n";
    }

    delete[] hTids;
    hipFree(dTids);

    return 0;
}
