//
// nvcc list_threads.cu
// 
// basic into to cuda kernel
//
#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>

using namespace std;

__global__
void saveTid(int *tids, int numElements) {
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid < numElements) {
        tids[tid*2] = blockIdx.x;
        tids[tid*2+1] = threadIdx.x;
    }
}

int main(int argc, char *argv[]) {

    if(argc < 3) {
        cout << "missing argument.\nUsage: list_threads <numElements> <numThreads>\n"
                "try: list_threads 20 5\n";
        return -1;
    }

    int numElements = atoi(argv[1]);
    int numThreads = atoi(argv[2]);
    int *dTids;
    
    int threadsPerBlock = numThreads;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    cout << "CUDA kernel launch with " << blocksPerGrid << " blocks of " << threadsPerBlock << " threads\n";

    hipMalloc(&dTids, sizeof(int)*numElements*2);
    saveTid<<<blocksPerGrid,threadsPerBlock>>>(dTids, numElements);
    
    int *hTids = new int[numElements * 2];
    
    hipMemcpy(hTids, dTids, sizeof(int) * numElements * 2, hipMemcpyDeviceToHost);

    for(int i = 0; i < numElements; ++i) {
        std::cout << i << ": blockId " << hTids[i*2] << ", threadId " << hTids[i*2+1] << "\n";
    }

    delete[] hTids;
    hipFree(dTids);

    return 0;
}
