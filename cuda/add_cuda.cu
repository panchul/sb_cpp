//
// $ nvcc add.cu -o add_cuda
// $ ./add_cuda
// Max error: 0.000000
//
#include <hip/hip_runtime.h>

#include <iostream>
#include <math.h>

using namespace std;

// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y, float *z, int batch_size)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  i *= batch_size;
  if(i<n) {
    int e = min(n, i + batch_size);
	  while ( i < e) {
      z[i] = x[i] + y[i];
      ++i;
    }
  }
}

int main(void) {

  int N = 1<<30;
  float *dx;
  float *dy;
  float *dz;

  // Allocate Unified Memory – accessible from CPU or GPU --- probably SLOW
  hipMallocManaged(&dx, N*sizeof(float));
  hipMallocManaged(&dy, N*sizeof(float));
  hipMallocManaged(&dz, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    dx[i] = i;
    dy[i] = -i;
  }

    int block_size = 4096;
    int threadsPerBlock = 4096;
    int blocksPerGrid =(N/block_size + threadsPerBlock - 1) / threadsPerBlock;
    cout << "CUDA kernel launch with " << blocksPerGrid << " blocks of " << threadsPerBlock << " threads\n";

    add<<<blocksPerGrid,threadsPerBlock>>>(N, dx, dy, dz, block_size);

  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 0.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(dz[i]-0.0f));
  std::cout << "Max error: " << maxError << std::endl;

  hipFree(dx);
  hipFree(dy);
  hipFree(dz);
  
  return 0;
}
